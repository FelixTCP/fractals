#include "hip/hip_runtime.h"
#include "color.hpp"
#include <string>
#include <iostream>
#include <chrono>
#include <fstream>

__global__ 
void draw_pixel(const double real_anker,
                const double imag_anker,
                const double real_step,
                const double imag_step,
                int *result,
                const int max_iter,
                const int width,
                const int height,
                const long long already_calculated_pixels){
  const int index = threadIdx.x + blockIdx.x * blockDim.x + already_calculated_pixels;

  if (index >= width * height)
    return;

  const double real = real_anker + real_step * (index % width);
  const double imag = imag_anker - imag_step * (index / height);
  
  int i = 0;
  double z_real = 0; 
  double z_imag = 0;
  double z_real_new = 0;
  double z_imag_new = 0;
  
  while (i < max_iter){
    z_real_new = z_real * z_real - z_imag * z_imag + real;
    z_imag_new = z_real * z_imag + z_imag * z_real + imag;

    if (z_real_new > 2 || z_real_new < -2 ||
        z_imag_new > 2 || z_imag_new < -2)
      break;

    z_real = z_real_new;
    z_imag = z_imag_new;
    i++;
  }
  result[index] = i;
}

int main(int argc, char **argv){
  auto start = std::chrono::high_resolution_clock::now();

  const int width = atoi(argv[1]);
  const int height = atoi(argv[2]);
  const std::string filename = std::string(argv[3]);
  png::image<png::rgb_pixel> image(width, height);

  const double anker_real = atof(argv[4]);
  const double max_real = atof(argv[5]);
  const double anker_imag = atof(argv[6]);
  const double min_imag = atof(argv[7]);
  const int max_iter = argc >= 9 ? atoi(argv[8]) : 2000;

  const long long pixels = width * height;

  int *result;
  result = (int *)malloc(pixels * sizeof(int));

  if (!result){auto end = std::chrono::high_resolution_clock::now();

    // Calculate the difference in seconds
    auto duration = std::chrono::duration_cast<std::chrono::seconds>(end - start);
    std::cerr << "Failed to allocate memory on the host." << std::endl;
    return -1;
  }

  const double step_real = (max_real - anker_real) / width;
  const double step_imag = (anker_imag - min_imag) / height;

  int *d_result;
  hipMalloc(&d_result, pixels * sizeof(int));

  int blockSize, gridSize;
  hipOccupancyMaxPotentialBlockSize(&blockSize, &gridSize, draw_pixel, 0, 0);

  // TODO strided loop 
  // TODO use fast maths flag??
  // TODO https://developer.nvidia.com/blog/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
  long long already_calculated_pixels = 0;
  
  hipEvent_t start_kernel, stop_kernel;
  float time_kernel;
  hipEventCreate(&start_kernel);
  hipEventCreate(&stop_kernel);
  hipDeviceSynchronize();
  hipEventRecord(start_kernel, 0);
  while (already_calculated_pixels < pixels){
    draw_pixel<<<gridSize, blockSize>>>(anker_real,
                                        anker_imag,
                                        step_real,
                                        step_imag,
                                        d_result,
                                        max_iter,
                                        width,
                                        height,
                                        already_calculated_pixels);
    already_calculated_pixels += gridSize * blockSize - 1;
  }
  hipDeviceSynchronize();

  // Calculate the difference in milliseconds and save it to kernel_time_i
  hipEventRecord(stop_kernel, 0);
  hipEventSynchronize(stop_kernel);
  hipEventElapsedTime(&time_kernel, start_kernel, stop_kernel);
  hipEventDestroy(start_kernel);
  hipEventDestroy(stop_kernel);
  std::ofstream out("kernel_time", std::ios::app);
  out << time_kernel << "\n";
  out.close();

  hipMemcpy(result, d_result, pixels * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_result);

  for (int i = 0; i < pixels; ++i){
    if (result[i] == max_iter)
      continue;
    image.set_pixel(i % width, i / height, color_map(result[i] % (max_iter / 2), max_iter / 2));
  }

  image.write(filename);

  if (argc >= 10 && std::string(argv[9]) == "-t"){
    auto diff = std::chrono::high_resolution_clock::now() - start;
    std::cout << "Time to generate frame: " << diff.count() << " seconds" << std::endl;
  }

  return 0;
}
